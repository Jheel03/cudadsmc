#include <stdio.h>
#include <string.h>
#include <initialization.h>
#include <translation_boundaryCollisions.h>
#include <indexing.h>
#include <binaryCollisions.h>
#include <memoryOps.cuh>

void posHostToDevice();
void posDeviceToHost();
void velHostToDevice();
void velDeviceToHost();

int main() {

	Setup();
	plot(1);

	for (unsigned int j = 2; j <= 200; j++) {

		translationInitialize();
		velDeviceToHost();
		plot(j);
		indexing();
		initializeBinaryCollisions();
		posHostToDevice();
		velHostToDevice();
		printf("\nIteration %u Completed.", j - 1);
	}
	

	return 0;

}





void posHostToDevice() {
	hipMemcpy(DevPosX, PosX, TotalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(DevPosY, PosY, TotalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void velHostToDevice() {
	hipMemcpy(DevVelX, VelX, TotalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(DevVelY, VelY, TotalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(DevVelZ, VelZ, TotalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void posDeviceToHost() {
	hipMemcpy(PosX, DevPosX, TotalSimulatedParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(PosY, DevPosY, TotalSimulatedParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

void velDeviceToHost() {
	hipMemcpy(VelX, DevVelX, TotalSimulatedParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(VelY, DevVelY, TotalSimulatedParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(VelZ, DevVelZ, TotalSimulatedParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}