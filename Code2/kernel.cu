#include "hip/hip_runtime.h"
#include <stdio.h>
#include <particleGenerator.cuh>
#include <fileOperations.h>
#include <memoryOps.cuh>
#include <string.h>
#include <boundaryCollisions.cuh>
#include <indexing.h>
#include <binaryCollisions.h>


int main() {

	hipError_t error;

	//////////////Initialization/////////////////
	initializeGasState();
	double *boundaryNodesX, *boundaryNodesY, *boundaryNormalsX, *boundaryNormalsY, *boundaryVectorsX, *boundaryVectorsY;
	unsigned int* indexArray;
	initializeDomain(&boundaryNodesX, &boundaryNodesY, &boundaryNormalsX, &boundaryNormalsY, &boundaryVectorsX, &boundaryVectorsY);
	unsigned int nWalls = 2 * (numberCellsX + numberCellsX);
	indexArray = (unsigned int*)malloc(numberCellsX * numberCellsY * sizeof(unsigned int));

	/////////////////////////////////////////////
	fileWriting(2 * (numberCellsX + numberCellsY), "plots/boundaryNodes.txt", boundaryNodesX, boundaryNodesY);
	fileWriting(2 * (numberCellsX + numberCellsY), "plots/boundaryVectors.txt", boundaryVectorsX, boundaryVectorsY);
	fileWriting(2 * (numberCellsX + numberCellsY), "plots/boundaryNormals.txt", boundaryNormalsX, boundaryNormalsY);
	
	/////////////Particles Generation//////////////
	double *velocityX, *velocityY, *velocityZ, *positionX, *positionY, *newPositionX, *newPositionY, *newVelocityX, *newVelocityY;
	particlesGeneratorInitialize(&velocityX, &velocityY, &velocityZ, &positionX, &positionY);
	///////////////////////////////////////////////
	fileWriting(totalSimulatedParticles, "plots/1.txt", positionX, positionY);

	/*for (unsigned int i = 0; i < totalSimulatedParticles; i++) {
		printf("\n%d: (%4e, %4e, %4e, %4e, %4e)", i, positionX[i], positionX[i], velocityX[i], velocityY[i], velocityZ[i]);
	}*/

	///////////////////Initializing Device Arrays//////////////////
	double *devPositionX, *devPositionY, *devVelocityX, *devVelocityY, *devVelocityZ;
	double *dPositionX, *dPositionY, *devDPositionX, *devDPositionY;
	double *devNewPositionX, *devNewPositionY;
	double *newDt;
	dPositionX = (double*)malloc(totalSimulatedParticles * sizeof(double));
	dPositionY = (double*)malloc(totalSimulatedParticles * sizeof(double));
	newPositionX = (double*)malloc(totalSimulatedParticles * sizeof(double));
	newPositionY = (double*)malloc(totalSimulatedParticles * sizeof(double));
	newVelocityX = (double*)malloc(totalSimulatedParticles * sizeof(double));
	newVelocityY = (double*)malloc(totalSimulatedParticles * sizeof(double));
	hipMalloc(&devPositionX, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devPositionY, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devDPositionX, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devDPositionY, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devVelocityX, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devVelocityY, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devVelocityZ, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devNewPositionX, totalSimulatedParticles * sizeof(double));
	hipMalloc(&devNewPositionY, totalSimulatedParticles * sizeof(double));
	hipMalloc(&newDt, totalSimulatedParticles * sizeof(double));

	hipMemcpy(devPositionX, positionX, totalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devPositionY, positionY, totalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devVelocityX, velocityX, totalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devVelocityY, velocityY, totalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devVelocityZ, velocityZ, totalSimulatedParticles * sizeof(double), hipMemcpyHostToDevice);

	double *devBoundaryVectorsX, *devBoundaryVectorsY, *devBoundaryNodesX,
		*devBoundaryNodesY, *devBoundaryNormalsX, *devBoundaryNormalsY;
	hipMalloc(&devBoundaryNodesX, nWalls * sizeof(double));
	hipMalloc(&devBoundaryNodesY, nWalls * sizeof(double));
	hipMalloc(&devBoundaryVectorsX, nWalls * sizeof(double));
	hipMalloc(&devBoundaryVectorsY, nWalls * sizeof(double));
	hipMalloc(&devBoundaryNormalsX, nWalls * sizeof(double));
	hipMalloc(&devBoundaryNormalsY, nWalls * sizeof(double));

	hipMemcpy(devBoundaryNodesX, boundaryNodesX, nWalls * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devBoundaryNodesY, boundaryNodesY, nWalls * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devBoundaryVectorsX, boundaryVectorsX, nWalls * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devBoundaryVectorsY, boundaryVectorsY, nWalls * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devBoundaryNormalsX, boundaryNormalsX, nWalls * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devBoundaryNormalsY, boundaryNormalsY, nWalls * sizeof(double), hipMemcpyHostToDevice);
	////////////////////////////////////////////////////////////////


	///////////////////Initializing BC Variables/////////////////////
	double *intersectionParameter, *minimumIntersectionParameter;
	unsigned int *intersectionFlag, *intersectionWallId, *tempIntersectionFlag, *outFlag, *devOutFlag;
	outFlag = (unsigned int*)malloc(totalSimulatedParticles * sizeof(unsigned int));
	hipMalloc(&intersectionParameter, nWalls * totalSimulatedParticles * sizeof(double));
	hipMalloc(&minimumIntersectionParameter, totalSimulatedParticles * sizeof(double));
	hipMalloc(&intersectionFlag, totalSimulatedParticles * sizeof(unsigned int));
	hipMalloc(&tempIntersectionFlag, totalSimulatedParticles * sizeof(unsigned int));
	hipMalloc(&intersectionWallId, totalSimulatedParticles * sizeof(unsigned int));
	hipMalloc(&devOutFlag, totalSimulatedParticles * sizeof(unsigned int));
	unsigned int *hostIntersectionFlag;
	unsigned int *devCounter;								//Here atomicAdd doesnot support unsigned long
	hipMalloc(&devCounter, sizeof(unsigned int));
	unsigned int *hostCollisionCounter;
	hostCollisionCounter = (unsigned int*)malloc(sizeof(unsigned int));
	hostIntersectionFlag = (unsigned int*)malloc(totalSimulatedParticles * sizeof(unsigned int));
	double *newVelocitiesN, *newVelocitiesNX, *newVelocitiesNY;
	////////////////////////////////////////////////////////////////


	char str1[50];
	char str3[10];
	char str2[10];

	dim3 blockSize;
	dim3 gridSize;

	////////////////Main Loop Starts///////////////
	for (unsigned int j = 2; j <= 200; j++) {

		boundaryCollisionsInitialize(&totalSimulatedParticles, positionX, positionY, velocityX, velocityY, velocityZ,
			devPositionX, devPositionY, devVelocityX, devVelocityY, devVelocityZ, intersectionFlag, tempIntersectionFlag,
			intersectionParameter, minimumIntersectionParameter, nWalls, devCounter, newDt, dt, &newVelocitiesN, &newVelocitiesNX, 
			&newVelocitiesNY, devDPositionX, devDPositionY, devBoundaryNodesX, devBoundaryNodesY, devBoundaryVectorsX, 
			devBoundaryVectorsY, devBoundaryNormalsX, devBoundaryNormalsY, intersectionWallId, j, hostCollisionCounter);

		indexingInitialization(&totalSimulatedParticles, numberCellsX, numberCellsY, cellHeight, cellLength, positionX, positionY, 
			velocityX, velocityY, newPositionX, newPositionY, indexArray, newVelocityX, newVelocityY);

		hipMemcpy(positionX, newPositionX, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);
		hipMemcpy(positionY, newPositionY, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);
		hipMemcpy(velocityX, newVelocityX, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);
		hipMemcpy(velocityY, newVelocityY, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);

		initializeBinaryCollisions(indexArray, velocityX, velocityY);
		hipMemcpy(velocityX, newVelocityX, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);
		hipMemcpy(velocityY, newVelocityY, totalSimulatedParticles * sizeof(double), hipMemcpyHostToHost);


		setString(str1, '\0', 50);
		setString(str2, '\0', 10);
		setString(str3, '\0', 10);
		assignString(str1, "plots/", 50);
		assignString(str3, ".txt", 10);
		_itoa_s(j, str2, 10);
		strcat(str1, str2);
		strcat(str1, str3);
		//printf("\n%s", str1);
		fileWriting(totalSimulatedParticles, str1, positionX, positionY);
		
	}

	/*hipFree(devPositionX);
	hipFree(devPositionY);
	hipFree(devDPositionX);
	hipFree(devDPositionY);
	hipFree(devVelocityX);
	hipFree(devVelocityY);
	hipFree(devVelocityZ);*/
	hipDeviceReset();

	free(positionX);
	free(positionY);
	free(velocityX);
	free(velocityY);
	free(velocityZ);
	free(dPositionX);
	free(dPositionY);
	free(boundaryNodesX);
	free(boundaryNodesY);
	free(boundaryNormalsX);
	free(boundaryNormalsY);
	free(boundaryVectorsX);
	free(boundaryVectorsY);
	free(newPositionX);
	free(newPositionY);
	free(newVelocityX);
	free(newVelocityY);
	free(indexArray);

	return 0;
}




